/*!
 * Functions to assist to identifying and characterizing CUDA devices available
 * \author Nathan A. Mahynski
 */

#include "cudaHelper.h"
#include <iostream>
using namespace std;

systemProps::systemProps (string fname) {
	output_ = new std::ofstream(fname.c_str());
}

systemProps::~systemProps () {
	if (output_ != &std::cout) {
		delete output_;
	}
}

/*!
 * Display properties of all CUDA capable devices
 */
__host__ void systemProps::displayCudaProps () {
	checkCudaDevices_();
	
	for (unsigned int i = 0; i < device_.size(); ++i) {
		*output_ << "# > Device Name: " <<  device_[i].name << endl;
		*output_ << "# -----------------------------------" << endl;
		*output_ << "# Total Global Memory: " << device_[i].totalGlobalMem/1024 << " KB" << endl;
		*output_ << "# Shared Memory available per Block: " << device_[i].sharedMemPerBlock/1024 << " KB" << endl;
		*output_ << "# Registers per Thread Block: " << device_[i].regsPerBlock << endl;
		*output_ << "# Warp Size: " << device_[i].warpSize << endl;
		*output_ << "# Memory Pitch: " << device_[i].memPitch << endl;
		*output_ << "# Maximum Threads per Block: " << device_[i].maxThreadsPerBlock << endl;
		*output_ << "# Maximum Thread Dimensions (Block): " << device_[i].maxThreadsDim[0] << "x" << device_[i].maxThreadsDim[1] << "x" << device_[i].maxThreadsDim[2] << endl;
		*output_ << "# Maximum Thread Dimensions (Grid): " << device_[i].maxGridSize[0] << "x" << device_[i].maxGridSize[1] << "x" << device_[i].maxGridSize[2] << endl;
		*output_ << "# Total Constant Memory: " << device_[i].totalConstMem << " bytes" << endl;
		*output_ << "# CUDA version: " << device_[i].major << "." << device_[i].minor << endl;
		*output_ << "# Clock Rate: " << device_[i].clockRate << " kHz" << endl;
		*output_ << "# Texture Alignment: " << device_[i].textureAlignment << " bytes"<< endl;
		if (device_[i].deviceOverlap == 0) {
			*output_ << "# Device Overlap: Not Allowed" << endl;
		} else {
			*output_ << "# Device Overlap: Allowed" << endl;
		}
		*output_ << "# Number of Multiprocessors: " << device_[i].multiProcessorCount << endl;
		*output_ << "# -----------------------------------"<< endl;
	}
}

/*!
 * Check CUDA devices and store all their properties
 */
__host__ void systemProps::checkCudaDevices_ () {
	int device_Count = 0;
	hipError_t error = hipGetDeviceCount(&device_Count);
	
	if (error != hipSuccess) {
		*output_ << "No CUDA capable devices found." << endl;
		device_Count = 0;
	}
	
	device_.resize(device_Count);
	
	for (unsigned int i = 0; i < device_Count; ++i) {
		hipGetDeviceProperties(&device_[i], i);
	}
}

/*!
 * Identify CPU host information
 */
void systemProps::checkHost_ () {
	struct utsname Uname;
	struct hostent *host;
	char ihname[64], ident[1024];
	
	if ((uname(&Uname) < 0) || (gethostname(ihname,64) != 0)) {
		hostname_ = "Unknown Hostname";
		return;
	}
	
	host=gethostbyname(ihname);
	strcpy(ident,Uname.sysname);
	strcat(ident," ");
	strcat(ident,Uname.release);
	strcat(ident," ");
	strcat(ident,Uname.machine);
	strcat(ident," [");
	strcat(ident,(*host).h_name);
	strcat(ident,"]");
	
	hostname_ = ident;
}
