#include "hip/hip_runtime.h"
/*!
 * Integration
 * \date 11/19/13
 */

#include "system.h"
#include <math.h>
#include "common.h"
#include "integrator.h"
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/system_error.h>
#include "potential.h"
#include <omp.h>

// Cuda Potential must be compiled together with the integrator so they are here, not in their own file

/*!
 * Compute the minimum image distance between two atoms on the GPU.
 *
 * \param [in] p1 Coordinates of atom 1
 * \param [in] p2 Coordinates of atom 2
 * \param [out] dr Vector pointing from atom1 to atom2
 * \param [in] box Coordinates of box
 */
__device__ float dev_pbcDist2 (const float3 *p1, const float3 *p2, float3 *dr, const float3 *box) {
	double d = 0.0;

	dr->x = p2->x - p1->x;
	while (dr->x > box->x/2.0) {
		dr->x -= box->x;
	}
	while (dr->x <= -box->x/2.0) {
		dr->x += box->x;
	}
	d += dr->x*dr->x;

	dr->y = p2->y - p1->y;
	while (dr->y > box->y/2.0) {
		dr->y -= box->y;
    	} 
	while (dr->y <= -box->y/2.0) {
		dr->y += box->y;
	}
	d += dr->y*dr->y;

	dr->z = p2->z - p1->z;
	while (dr->z > box->z/2.0) {
		dr->z -= box->z;
	}
	while (dr->z <= -box->z/2.0) {
		dr->z += box->z;
	}
	d += dr->z*dr->z;

	return d;
}

/*!
 * An arbitrary potential that could be changed to meet the user's needs in the future.
 */
__device__ float dev_pairUF (const float3 *p1, const float3 *p2, float3 *pairForce, const float3 *box, const float *args, const float *rcut) {
        return 0.0;
}

/*!
 * Pairwise interaction between 2 atoms (shifted lennard-jones)
 *
 * \param [in] p1 Pointer to atom 1's position
 * \param [in] p2 Pointer to atom 2's position
 * \param [in, out] pairForce Force atom 1 experiences due to atom 2
 * \param [in] box Pointer to box dimensions
 * \param [in] args Additional arguments, in this case {epsilon, sigma, delta, ushift}
 * \param [in] rcut Cutoff distance; NOTE: this MUST already incorporate the delta shift, ie. if r < ((rc' + delta) = rc), then force is computed
*/
__device__ float dev_slj (const float3 *p1, const float3 *p2, float3 *pairForce, const float3 *box, const float *args, const float *rcut) {
    float3 dr;
	float r2 = dev_pbcDist2(p1, p2, &dr, box);

	// check that r > delta and throw/catch
	if (r2 <= args[2]*args[2]) {
		// shift to just past the singularity and allow to run
		r2 = 1.0001*args[2]*args[2];
	}
	
	if (r2 < (*rcut)*(*rcut)) {
		float r = sqrt(r2);
        float x = r - args[2];
		float b = 1.0/x, a = args[1]*b, a2 = a*a, a6 = a2*a2*a2, factor;
		factor = 24.0*args[0]*a6*(2.0*a6-1.0)*b/r;
		pairForce->x = -factor*dr.x;
		pairForce->y = -factor*dr.y;
		pairForce->z = -factor*dr.z;
		return 4.0*args[0]*(a6*a6-a6)+args[3];
	} else {
		pairForce->x = 0.0;
		pairForce->y = 0.0;
		pairForce->z = 0.0;
		return 0.0;
	}
}

/*!
 * From the host, call this kernel to loop over each atom's neighbor list
 *
 * \param [in] dev_atoms Atoms (sys.atoms)
 * \param [in] nlist Neighbor list of all atoms indexed in a linear array
 * \param [in] nlist_index Index to start at in nlist to find an atom's neighbors
 * \param [out] force Net force each atom experiences
 * \param [in] Box dimensions
 * \param [out] Up_each Potential energy of each atom in the field, the total is this summed divided by 2
 * \param [in] natoms Number of atoms
 * \param [in] args Pair potential arguments
 * \param [in] rcut Cutoff distance for potential
 * \param [in] pFlag Flags which potential function to use
 */
__global__ void loopOverNeighbors (atom* dev_atoms, int* nlist, int* nlist_index, float3* force, float3* box, float* Up_each, int* natoms, float* args, float* rcut, int* pFlag) {
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (tid < *natoms) {
		float Up = 0.0;
		const int start = nlist_index[tid];
		
		// choose the potential function
		pointFunction_t pFunc;
		if (*pFlag == 0) {
			pFunc = dev_slj;
		} else {
			pFunc = dev_pairUF;
		}

		float3 myforce;
		myforce.x = 0;
		myforce.y = 0;
		myforce.z = 0;

		// loop over this atom's neighbors
		for (unsigned int i = start+1; i < start+1+nlist[start]; ++i) {
			// compute potential between dev_atoms[nlist[i]] and dev_atoms[tid]
			float3 dummyForce;
			Up += pFunc (&dev_atoms[nlist[i]].pos, &dev_atoms[tid].pos, &dummyForce, box, args, rcut);

			myforce.x += dummyForce.x;
			myforce.y += dummyForce.y;
			myforce.z += dummyForce.z;	
		}

		// maintaining sign convention
		force[tid].x = -myforce.x;
		force[tid].y = -myforce.y;
		force[tid].z = -myforce.z;
		Up_each[tid] = Up;
	}	
}

/*!
 * Calculate the pairwise forces in a system.  This also calculates the potential energy of a system.
 * The kinetic energy is calculated during the verlet integration.
 *
 * \param [in, out] sys System definition
 */
void integrator::calcForce (systemDefinition &sys) {
	float Up = 0.0;
	const float invMass = 1.0/sys.mass();

	// set which potential to use
	std::vector < int > pFlag (1, -1);
	if (sys.potential == dev_slj) {
		pFlag[0] = 0;
	} else if (sys.potential == dev_pairUF) {
		pFlag[0] = 1;
	} else {
		throw customException ("Cannot understand which potential function to use");
	}

	thrust::device_vector < int > dev_pFlag (pFlag.begin(), pFlag.end());
	int* dev_pFlag_ptr = thrust::raw_pointer_cast(&dev_pFlag[0]);

	// box dimensions
	std::vector < float3 > box (1, sys.box());
	thrust::device_vector < float3 > sysbox (box.begin(), box.end());
	float3* dev_sysbox_ptr = thrust::raw_pointer_cast(&sysbox[0]);

	// check update for neighborlist
	cl_.checkUpdate(sys);
	
	// number of atoms
	thrust::device_vector < int > dev_natoms(1, sys.numAtoms());
	int* dev_natoms_ptr = thrust::raw_pointer_cast(&dev_natoms[0]);

	// copy sys.atoms to GPU
	thrust::device_vector < atom > dev_atoms (sys.atoms.begin(), sys.atoms.end());
	atom* dev_atoms_ptr = thrust::raw_pointer_cast(&dev_atoms[0]);

	// copy neighborlists to GPU
	thrust::device_vector < int > dev_neighbor_list (cl_.nlist.begin(), cl_.nlist.end());
	int* dev_neighbor_list_ptr = thrust::raw_pointer_cast(&dev_neighbor_list[0]);

	// copy location of where the neighbors for each atoms starts
	thrust::device_vector < int > dev_neighbor_index (cl_.nlist_index.begin(), cl_.nlist_index.end());
	int* dev_neighbor_index_ptr = thrust::raw_pointer_cast(&dev_neighbor_index[0]);
	
	// create acc and Up to store results in
	thrust::device_vector < float3 > dev_force (sys.numAtoms());
	float3* dev_force_ptr = thrust::raw_pointer_cast(&dev_force[0]);
	thrust::device_vector < float > dev_Up_each_atom (sys.numAtoms());
	float* dev_Up_each_atom_ptr = thrust::raw_pointer_cast(&dev_Up_each_atom[0]);
	
	// potential arguments
	std::vector < float > potArgs = sys.potentialArgs();
	std::vector < float > rcut (1, sys.rcut());
	thrust::device_vector < float > dev_args (potArgs.begin(), potArgs.end());
	float* dev_args_ptr = thrust::raw_pointer_cast(&dev_args[0]);
	thrust::device_vector < float > dev_rcut (rcut.begin(), rcut.end());
	float* dev_rcut_ptr = thrust::raw_pointer_cast(&dev_rcut[0]);

	// invoke kernel to compute
	loopOverNeighbors <<< sys.cudaBlocks, sys.cudaThreads >>> (dev_atoms_ptr, dev_neighbor_list_ptr, dev_neighbor_index_ptr, dev_force_ptr, dev_sysbox_ptr, dev_Up_each_atom_ptr, dev_natoms_ptr, dev_args_ptr, dev_rcut_ptr, dev_pFlag_ptr);
	
	// call a reduction to collect Up then divide by 2 since double counted
	Up = thrust::reduce(dev_Up_each_atom.begin(), dev_Up_each_atom.end(), (float) 0.0, thrust::plus<float>());
	Up /= 2.0;	// pairs are double counted
	
	// store accelerations on atoms	
	std::vector < float3 > netForces (sys.numAtoms());
	thrust::copy(dev_force.begin(), dev_force.end(), netForces.begin());

	#pragma omp parallel for schedule(dynamic,OMP_CHUNK)
	for (unsigned int i = 0; i < sys.numAtoms(); ++i) {
		sys.atoms[i].acc.x = netForces[i].x*invMass;
		sys.atoms[i].acc.y = netForces[i].y*invMass;
		sys.atoms[i].acc.z = netForces[i].z*invMass;
	}	

	// set Up
	sys.setPotE(Up);
}
