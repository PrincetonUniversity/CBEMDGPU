#include "hip/hip_runtime.h"
/*!
 * Integration
 * \author Nathan A. Mahynski
 * \date 11/19/13
 */

#include "system.h"
#include <exception>
#include <math.h>
#include "common.h"
#include "integrator.h"
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

/*!
 * From the host, call this kernel to loop over each atom's neighbor list
 *
 * \param [in] dev_atoms Atoms (sys.atoms)
 * \param [in] nlist Neighbor list of all atoms indexed in a linear array
 * \param [in] nlist_index Index to start at in nlist to find an atom's neighbors
 * \param [out] force Net force each atom experiences
 * \param [out] Up Potential energy of each atom in the field, the total is this summed divided by 2
 * \param [in] natoms Number of atoms
 */
__global__ void loopOverNeighbors (atom* dev_atoms, int* nlist, int* nlist_index, float3* force, float* Up_each, int* natoms) {
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < *natoms) {
		const int nn = nlist_index[tid]; // number of neighbors
		for (unsigned int i = nlist_index[tid]+1; i < nlist_index[tid]+1+nn; ++i) {
			// compute potential between dev_atoms[nlist[i]] and dev_atoms[tid]
			
		}
	}	
}

/*!
 * Calculate the pairwise forces in a system.  This also calculates the potential energy of a system.
 * The kinetic energy is calculated during the verlet integration.
 *
 * \param [in, out] sys System definition
 */
void integrator::calcForce (systemDefinition &sys) {
	float Up = 0.0;
	const float3 box = sys.box();
	const float invMass = 1.0/sys.mass();
	
	// check update for neighborlist
	cl_.checkUpdate(sys);

	// number of atoms
	thrust::device_vector < int > dev_natoms(1, sys.numAtoms());
	int* dev_natoms_ptr = thrust::raw_pointer_cast(&dev_natoms[0]);

	// copy sys.atoms to GPU
	thrust::device_vector < atom > dev_atoms (sys.atoms.begin(), sys.atoms.end());
	atom* dev_atoms_ptr = thrust::raw_pointer_cast(&dev_atoms[0]);

	// copy neighborlists to GPU
	thrust::device_vector < int > dev_neighbor_list (cl_.nlist.begin(), cl_.nlist.end());
	int* dev_neighbor_list_ptr = thrust::raw_pointer_cast(&dev_neighbor_list[0]);

	// copy location of where the neighbors for each atoms starts
	thrust::device_vector < int > dev_neighbor_index (cl_.nlist_index.begin(), cl_.nlist_index.end());
	int* dev_neighbor_index_ptr = thrust::raw_pointer_cast(&dev_neighbor_index[0]);
	
	// create acc and Up to store results in
	thrust::device_vector < float3 > dev_force (sys.numAtoms());
	float3* dev_force_ptr = thrust::raw_pointer_cast(&dev_force[0]);
	thrust::device_vector < float > dev_Up_each_atom (sys.numAtoms());
	float* dev_Up_each_atom_ptr = thrust::raw_pointer_cast(&dev_Up_each_atom[0]);

	// invoke kernel to compute
	loopOverNeighbors <<< sys.cudaBlocks, sys.cudaThreads >>> (dev_atoms_ptr, dev_neighbor_list_ptr, dev_neighbor_index_ptr, dev_force_ptr, dev_Up_each_atom_ptr, dev_natoms_ptr);

	// call a reduction to collect Up then divide by 2 since double counted
	Up = thrust::reduce(dev_Up_each_atom.begin(), dev_Up_each_atom.end(), (float) 0.0, thrust::plus<float>());
	Up /= 2.0;	// pairs are double counted

	// store accelerations on atoms	
	std::vector < float3 > netForces (sys.numAtoms());
	thrust::copy(dev_force.begin(), dev_force.end(), netForces.begin());
	for (unsigned int i = 0; i < sys.numAtoms(); ++i) {
		sys.atoms[i].acc.x = netForces[i].x*invMass;
		sys.atoms[i].acc.y = netForces[i].y*invMass;
		sys.atoms[i].acc.z = netForces[i].z*invMass;
	}	

	// set Up
	sys.setPotE(Up);
}
